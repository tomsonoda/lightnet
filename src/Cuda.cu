#include <stdio.h>

namespace gpu_cuda {

__device__ void cudaMakeArray(float *gpu_array, int N )
{
  hipMalloc(&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

__device__ void cudaPutArray( float *gpu_array, float *cpu_array, int N )
{
  hipMemcpy(gpu_array, cpu_array, N*sizeof(float), hipMemcpyHostToDevice);
}

__device__ void cudaGetArray( float *cpu_array, float *gpu_array, int N )
{
  hipMemcpy(cpu_array, gpu_array, N*sizeof(float), hipMemcpyDeviceToHost);
}

} // namespace gpu
