
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace gpu_cuda {

void cudaMakeArray(float *gpu_array, int N )
{
  hipMalloc(&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

void cudaPutArray( float *gpu_array, float *cpu_array, int N )
{
  hipMemcpy(gpu_array, cpu_array, N*sizeof(float), hipMemcpyHostToDevice);
}

void cudaGetArray( float *cpu_array, float *gpu_array, int N )
{
  hipMemcpy(cpu_array, gpu_array, N*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaClearArray( float *gpu_array, int N ){
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

__global__ void cudaAddArray(float * dz_next_layer, float *dz_in)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  dz_in[id] += dz_next_layer[id];
}

} // namespace gpu
