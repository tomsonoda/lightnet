#include "hip/hip_runtime.h"
#include <stdio.h>

#include "CudaObject.h"

namespace gpu_cuda {

__device__ unsigned int Rand(unsigned int randx)
{
  randx = randx*1103515245+12345;
  return randx&2147483647;
}

float *cudaMakeArray( int N )
{
  float *gpu_array;
  hipMalloc((void **)&gpu_array, N*sizeof(float));
  // hipMemset(&gpu_array, 0, N*sizeof(float));
  return gpu_array;
}

void cudaPutArray( float *gpu_array, float *cpu_array, int N )
{
  hipMemcpy(gpu_array, cpu_array, N*sizeof(float), hipMemcpyHostToDevice);
}

void cudaGetArray( float *cpu_array, float *gpu_array, int N )
{
  hipMemcpy(cpu_array, gpu_array, N*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaClearArray( float *gpu_array, int N )
{
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

__global__ void setRandom(float *gpu_array, int maxval )
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  gpu_array[id] = 1.0f / maxval * Rand(id) / float( RAND_MAX );
}

void cudaMakeRandomArray(float *gpu_array, int N, int maxval )
{
  hipMalloc(&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
  CudaObject cuda = CudaObject();
  dim3 grid = cuda.cudaGridSize(N);
  setRandom<<<grid, BLOCK>>>( gpu_array, maxval );
}

} // namespace gpu
