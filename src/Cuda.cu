#include "hip/hip_runtime.h"
#include <stdio.h>

#include "CudaObject.h"

namespace gpu_cuda {

__device__ unsigned int Rand(unsigned int randx)
{
  randx = randx*1103515245+12345;
  return randx&2147483647;
}

float *cudaMakeArray( int N )
{
  float *gpu_array;
  hipMalloc((void **)&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
  return gpu_array;
}

void cudaCheckError(hipError_t status)
{
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess){
        const char *s = hipGetErrorString(status);
        char buffer[256];
        printf("CUDA Error: %s\n", s);
        assert(0);
        snprintf(buffer, 256, "CUDA Error: %s", s);
        error(buffer);
    }
    if (status2 != hipSuccess){
        const char *s = hipGetErrorString(status);
        char buffer[256];
        printf("CUDA Error Prev: %s\n", s);
        assert(0);
        snprintf(buffer, 256, "CUDA Error Prev: %s", s);
        error(buffer);
    }
}

void cudaPutArray( float *gpu_array, float *cpu_array, int N )
{
  hipError_t status = hipMemcpy(gpu_array, cpu_array, N*sizeof(float), hipMemcpyHostToDevice);
  check_error(status);
}

void cudaGetArray( float *cpu_array, float *gpu_array, int N )
{
  hipError_t status = hipMemcpy(cpu_array, gpu_array, N*sizeof(float), hipMemcpyDeviceToHost);
  check_error(status);
}

void cudaClearArray( float *gpu_array, int N )
{
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

__global__ void setRandom(float *gpu_array, int maxval )
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  gpu_array[id] = 1.0f / maxval * Rand(id) / float( RAND_MAX );
}

void cudaMakeRandomArray(float *gpu_array, int N, int maxval )
{
  hipMalloc(&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
  CudaObject cuda = CudaObject();
  dim3 grid = cuda.cudaGridSize(N);
  setRandom<<<grid, BLOCK>>>( gpu_array, maxval );
}

} // namespace gpu
