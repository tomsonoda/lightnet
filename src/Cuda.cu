
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace gpu_cuda {

void cudaMakeArray(float *gpu_array, int N )
{
  hipMalloc(&gpu_array, N*sizeof(float));
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

void cudaPutArray( float *gpu_array, float *cpu_array, int N )
{
  hipMemcpy(gpu_array, cpu_array, N*sizeof(float), hipMemcpyHostToDevice);
}

void cudaGetArray( float *cpu_array, float *gpu_array, int N )
{
  hipMemcpy(cpu_array, gpu_array, N*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaClearArray( float *gpu_array, int N ){
  hipMemset(&gpu_array, 0, N*sizeof(float));
}

} // namespace gpu
