
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace gpu_cuda {

__global__

void calc(int n, float *x, float *y)
{
  for( int i = 0; i < n; ++i ){
    float v = x[i];
    if ( v < 0 ){
      v = 0.1 * v;
    }
    y[i] = v;
  }
}

void leakyReluForwardGPU(float *data_in, float *data_out, int N)
{
  float *d_in, *d_out;
  hipMalloc(&d_in,  N*sizeof(float));
  hipMalloc(&d_out, N*sizeof(float));

  hipMemcpy(d_in,  data_in,  N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, data_out, N*sizeof(float), hipMemcpyHostToDevice);

  const int threads_per_block = 2*32;
  const int blocks_per_grid = 1*12;
  calc<<<blocks_per_grid, threads_per_block>>>(N, d_in, d_out);

  hipMemcpy(data_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++){
    printf("data_size=%d pair: %f %f\n", N, data_in, data_out);
  }
}

} // namespace gpu
