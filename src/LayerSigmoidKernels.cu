#include "hip/hip_runtime.h"
#include <stdio.h>
#include "CudaObject.h"

namespace gpu_cuda {

__global__ void calcSigmoidForwardGPU(float *in, float *out, int elements)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  if( id < elements ){
    float v = in[id];
    v = 1.0f / (1.0f + exp( -v )); // sigmoid
    out[id] = v;
  }

  /* original
  for ( int i = 0; i < in_total_size; ++i ){
    out.data[i] = activator_function(in.data[i]);
  }
  */
}

__global__ void calcSigmoidBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *in, int elements )
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

  if( id < elements ){
    float x = dz_in[id] += dz_next_layer[id];
    float sig = 1.0f / (1.0f + exp( -x ));
    dz[id] +=  ( sig * (1 - sig) ) * dz_in[id]; // sigmoid_derivative * dz_in
  }

  /* original
  for( int i = 0; i < dz_in.size.b * dz_in.size.x * dz_in.size.y * dz_in.size.z; ++i ){
    dz_in.data[i] += dz_next_layer.data[i];
  }

  for ( int i = 0; i < in_total_size; ++i ){
    dz.data[i] += activator_derivative( in.data[i] ) * dz_in.data[i];
  }
  */
}

void sigmoidForwardGPU(float *in, float *out, int N)
{
  CudaObject cuda = CudaObject();
  dim3 grid = cuda.cudaGridSize(N);
  calcSigmoidForwardGPU<<<grid, BLOCK>>>(in, out, N);
}

void sigmoidBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *in, int N )
{
  CudaObject cuda = CudaObject();
  dim3 grid = cuda.cudaGridSize(N);
  calcSigmoidBackwardGPU<<<grid, BLOCK>>>( dz_next_layer, dz_in, dz, in, N );
}

} // namespace gpu
