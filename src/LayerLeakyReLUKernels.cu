#include "hip/hip_runtime.h"
#include <stdio.h>
#define BLOCK 512

namespace gpu_cuda {

dim3 cuda_gridsize(size_t n){
  size_t k = (n-1) / BLOCK + 1;
  size_t x = k;
  size_t y = 1;
  if(x > 65535){
      x = ceil(sqrt(k));
      y = (n-1)/(x*BLOCK) + 1;
  }
  //printf("%ld %ld %ld %ld\n", n, x, y, x*y*BLOCK);
  return {x, y, 1};
}

void calc(float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float v = x[i];
  if ( v < 0 ){
    v = 0.1 * v;
  }
  y[i] = v;
}

__global__ void leakyReluForwardGPU(float *data_in, float *data_out, int N)
{
  float *d_in, *d_out;
  hipMalloc(&d_in,  N*sizeof(float));
  hipMalloc(&d_out, N*sizeof(float));

  hipMemcpy(d_in,  data_in,  N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, data_out, N*sizeof(float), hipMemcpyHostToDevice);

  calc<<<cuda_gridsize(N), BLOCK>>>(d_in, d_out);

  hipMemcpy(data_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);
}

} // namespace gpu
