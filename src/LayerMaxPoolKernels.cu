#include "hip/hip_runtime.h"
#include <stdio.h>
#include "CudaObject.h"
#include "CudaCommon.cuh"

namespace gpu_cuda {

__global__ void calcMaxPoolForwardGPU(
  float *in,float *out,
  int in_size_x, int in_size_y, int in_size_z,
  int out_size_x, int out_size_y, int out_size_z,
  int stride, int kernel_size
)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int id_out = id;

  int x = id % out_size_x;
  id /= out_size_x;
  int y = id % out_size_y;
  id /= out_size_y;
  int z = id % out_size_z;
  id /= out_size_z;
  int b = id;

  int mapped_x = x * stride;
  int mapped_y = y * stride;

  float mval = -100000.0;
  for ( int j = 0; j < kernel_size; ++j ){
    for ( int i = 0; i < kernel_size; ++i ){

      int id_in = b * (in_size_z * in_size_x * in_size_y) +
        z * (in_size_x * in_size_y) +
        (mapped_x + i) * (in_size_x) +
        (mapped_y + j);

      float v = in[id_in];
      if ( v > mval ){
        mval = v;
      }
    }
  }
  out[id_out] = mval;
}

__global__ void calcMaxPoolBackwardGPU( float *dz_in, float *dz, float *in, float *out, int dz_size_x, int dz_size_y, int dz_size_z, int dz_in_size_x, int dz_in_size_y, int dz_in_size_z, int kernel_size, int stride ){
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int id_dz = id;

  int x = id % dz_size_x;
  id /= dz_size_x;
  int y = id % dz_size_y;
  id /= dz_size_y;
  int z = id % dz_size_z;
  id /= dz_size_z;
  int b = id;

  range_t rn = map_to_output( x, y, dz_in_size_x, dz_in_size_y, kernel_size, stride );

  float sum_error = 0;
  float in_value = in[id_dz];
  for ( int j = rn.min_y; j <= rn.max_y; ++j ){
    for ( int i = rn.min_x; i <= rn.max_x; ++i ){

      int out_index = (b * (dz_in_size_x * dz_in_size_y * dz_in_size_z) +
        z * (dz_in_size_x * dz_in_size_y) +
        j * (dz_in_size_x) +
        i );

      int is_max = in_value == out[out_index] ? 1 : 0;
      sum_error += is_max * dz_in[out_index];
    }
  }
  dz[id_dz] += sum_error;
  /*
  for ( int b = 0; b < in.size.b; ++b ){
    for ( int y = 0; y < in.size.y; ++y ){
      for ( int x = 0; x < in.size.x; ++x ){
        range_t rn = map_to_output( x, y );
        for ( int z = 0; z < in.size.z; ++z ){
          float sum_error = 0;
          float in_value = in( b, x, y, z );
          for ( int j = rn.min_y; j <= rn.max_y; ++j ){
            for ( int i = rn.min_x; i <= rn.max_x; ++i ){
              int is_max = in_value == out( b, i, j, z ) ? 1 : 0;
              sum_error += is_max * dz_in( b, i, j, z );
            }
          }
          dz( b, x, y, z ) += sum_error;
        }
      }
    }
  }
  */
}

void maxPoolForwardGPU(float *in, float *out, int in_size_x, int in_size_y, int in_size_z, int out_size_b, int out_size_x, int out_size_y, int out_size_z, int kernel_size, int stride )
{
  int out_N = out_size_b * out_size_x * out_size_y * out_size_z;
  CudaObject cuda = CudaObject();
  dim3 grid = cuda.cudaGridSize(out_N);
  calcMaxPoolForwardGPU<<<grid, BLOCK>>>(in, out, in_size_x, in_size_y, in_size_z, out_size_x, out_size_y, out_size_z, stride, kernel_size);
}

void maxPoolBackwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *in, float *out, int batch_size, int dz_size_x, int dz_size_y, int dz_size_z, int dz_in_size_x, int dz_in_size_y, int dz_in_size_z, int kernel_size, int stride )
{
  int in_N = batch_size * dz_in_size_x * dz_in_size_y * dz_in_size_z;
  CudaObject cuda = CudaObject();
  dim3 grid_in = cuda.cudaGridSize(in_N);
  cudaAddFirstArrayToSecondArray<<<grid_in, BLOCK>>>( dz_next_layer, dz_in );

  int N = batch_size * dz_size_x * dz_size_y * dz_size_z;
  dim3 grid = cuda.cudaGridSize(N);
  calcMaxPoolBackwardGPU<<<grid, BLOCK>>>( dz_in, dz, in, out, dz_size_x, dz_size_y, dz_size_z, dz_in_size_x, dz_in_size_y, dz_in_size_z, kernel_size, stride );
}

} // namespace gpu
