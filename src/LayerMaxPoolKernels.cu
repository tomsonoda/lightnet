#include "hip/hip_runtime.h"
#include <stdio.h>

namespace gpu_cuda {

__global__ void calcMaxPoolForwardGPU(float *in,float *out,
  int in_size_x, int in_size_y, int in_size_z,
  int size_x, int size_y, int size_z,
  int stride, int kernel_size)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int id_out = id;

  int x = id % size_x;
  id /= size_x;
  int y = id % size_y;
  id /= size_y;
  int z = id % size_z;
  id /= size_z;
  int b = id;

  int mapped_x = x * stride;
  int mapped_y = y * stride;
  float mval = -100000.0;
  for ( int j = 0; j < kernel_size; ++j ){
    for ( int i = 0; i < kernel_size; ++i ){

      int index =
      b * (in_size_z * in_size_x * in_size_y) +
      z * (in_size_x * in_size_y) +
      (mapped_x + i) * (in_size_x) +
      (mapped_y);

      float v = in[index];
      if ( v > mval ){
        mval = v;
      }
    }
  }
  out[id_out] = mval;
}

__global__ void calcMaxPoolBackwardGPU(float *in1, float *in2, float *in3, float* out)
{
}

void maxPoolForwardGPU(float *data_in, float *data_out,
  int in_size_b, int in_size_x, int in_size_y, int in_size_z,
  int out_size_b, int out_size_x, int out_size_y, int out_size_z,
  int stride, int kernel_size)
{
  float *d_in, *d_out;
  int in_N = in_size_b * in_size_x * in_size_y * in_size_z;
  int N = out_size_b * out_size_x * out_size_y * out_size_z;
  hipMalloc(&d_in,  in_N*sizeof(float));
  hipMalloc(&d_out, N*sizeof(float));
  hipMemcpy(d_in,  data_in,  in_N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_out, data_out, N*sizeof(float), hipMemcpyHostToDevice);
  dim3 grid = cudaGridSize(N);
  calcMaxPoolForwardGPU<<<grid, BLOCK>>>(d_in, d_out, in_size_x, in_size_y, in_size_z, out_size_x, out_size_y, out_size_z, stride, kernel_size);
  hipMemcpy(data_out, d_out, N*sizeof(float), hipMemcpyDeviceToHost);
}

void maxPoolBackwardGPU(float *data_in1, float *data_in2, float *data_in3, float *data_out, int N)
{
}

} // namespace gpu
