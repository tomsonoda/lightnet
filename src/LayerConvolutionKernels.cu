#include "hip/hip_runtime.h"
#include <stdio.h>
#include "CudaObject.h"
#include "CudaCommon.cuh"

namespace gpu_cuda {

__global__ void calcConvolutionForwardPaddedInGPU( float *in, float *padded_in,
    int in_size_x, int in_size_y, int in_size_z, int padding)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int id_out = id;

  int x = id % in_size_x;
  id /= in_size_x;
  int y = id % in_size_y;
  id /= in_size_y;
  int z = id % in_size_z;
  id /= in_size_z;
  int b = id;

  if( (x - padding ) > 0 && ( y - padding ) > 0){
    int in_index = (b * (in_size_z * in_size_x * in_size_y) +
    z * (in_size_x * in_size_y) +
    (y - padding) * (in_size_x) +
    (x - padding) );
    padded_in[id_out] = in[in_index];
  }
  /*
  for ( int b = 0; b < in.size.b; ++b ){
    for ( int z = 0; z < in.size.z; ++z ){
      for ( int y = 0; y < in.size.y; ++y ){
        for ( int x = 0; x < in.size.x; ++x ){
          padded_in( b, padding+x, padding+y, z ) = in( b, x, y, z );
        }
      }
    }
  }
  */
}

__global__ void calcConvolutionForwardGPU( float *out, float *padded_in, float *filters, int padded_in_size_x, int padded_in_size_y, int padded_in_size_z, int out_size_x, int out_size_y, int out_size_z, int kernel_size, int stride, int filter_size)
{
  int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  int id_out = id;

  int x = id % out_size_x;
  id /= out_size_x;
  int y = id % out_size_y;
  id /= out_size_y;
  int filter = id % out_size_z;
  id /= out_size_z;
  int b = id;

  int mapped_x = x * stride;
  int mapped_y = y * stride;

  float sum = 0.0;
  for ( int z = 0; z < padded_in_size_z; ++z ){
    for ( int j = 0; j < kernel_size; ++j ){
      for ( int i = 0; i < kernel_size; ++i ){
        int filter_index = z * (kernel_size * kernel_size) + j * kernel_size + i;
        int padded_in_index = b * (padded_in_size_x * padded_in_size_y * padded_in_size_z) + z * (padded_in_size_x * padded_in_size_y) + (mapped_y + j) * (padded_in_size_x) + (mapped_x + i);
        sum += filters[filter * filter_size + filter_index] * padded_in[padded_in_index];
      }
    }
  }
  out[id_out] = sum;

  /*
  for ( int b = 0; b < in.size.b; ++b ){
    int filters_size = filters.size();
    for ( int filter = 0; filter < filters_size; ++filter ){
      TensorObject<float> filter_data = filters[filter];
      for ( int y = 0; y < out.size.y; ++y ){
        for ( int x = 0; x < out.size.x; ++x ){
          TensorCoordinate mapped = map_to_input( { 0, (uint16_t)x, (uint16_t)y, 0 }, 0 );
          float sum = 0;
          for ( int z = 0; z < in.size.z; ++z ){
            for ( int j = 0; j < kernel_size; ++j ){
              for ( int i = 0; i < kernel_size; ++i ){
                sum += filter_data( 0, i, j, z ) * padded_in( b, mapped.x + i, mapped.y + j, z );
              }
            }
          }
          out( b, x, y, filter ) = sum;
        }
      }
    }
  }*/
}

__global__ void calcConvolutionBackwardGPU(float *in1, float *in2, float *in3, float* out)
{
/*
int k_end = filter_grads.size();
int kernel_size_2 = kernel_size * kernel_size;
int i_end = kernel_size_2 * in.size.z;
for ( int k = 0; k < k_end; ++k ){
  for ( int i = 0; i < i_end ; ++i ){
      filter_grads[k].data[i].grad = 0;
  }
}

int z_max = (int)filters.size();
std::vector< std::future<int> > thread_results;

for ( int b = 0; b < in.size.b; ++b ){

  thread_results.emplace_back(thread_pool.enqueue([&, b] {

    // code optimization.
    int dz_in_xy = dz_in.size.y * dz_in.size.x;
    int b_dz_in_xyz = b * dz_in.size.z * dz_in_xy;
    int padded_in_xy = padded_in.size.y * padded_in.size.x;
    int b_padded_in_xyz = b * padded_in.size.z * padded_in_xy;

    int y_end = padded_in.size.y - padding;
    for ( int y = 0; y < y_end; ++y ){

      int x_end = padded_in.size.x - padding;
      for ( int x = 0; x < x_end; ++x ){

        tensor_range_t rn = map_to_output( x, y );

        for ( int z = 0; z < in.size.z; ++z ){

          float sum = 0;
          // float padded_in_value = padded_in( b, x, y, z );
          float padded_in_value = padded_in.data[( b_padded_in_xyz ) + (z * padded_in_xy) + (y * padded_in.size.x) + x];

          for ( int j = rn.min_y; j <= rn.max_y; ++j ){
            int y_miny = y - j * stride;

            for ( int i = rn.min_x; i <= rn.max_x; ++i ){
              int x_minx = x - i * stride;

              int xyz = z * kernel_size_2 + y_miny * kernel_size + x_minx; // ( 0, x_minx, y_miny, z )

              for ( int k = 0; k < z_max; ++k ){
                // float d = dz_in( b, i, j, k );
                float d = dz_in.data[ b_dz_in_xyz + (k * dz_in_xy) + (j * dz_in.size.x) + i];
                // sum += filters[k].get( 0, x_minx, y_miny, z ) * d;
                sum += filters[k].data[xyz] * d;
                // filter_grads[k].get( 0, x_minx, y_miny, z ).grad += padded_in_value * d;
                filter_grads[k].data[xyz].grad += padded_in_value * d;
              }
            }
          }

          if( x>=padding && y>=padding ){
            dz( b, x - padding, y - padding, z ) += sum;
          }
        }

      }
    }
*/
}

void convolutionForwardGPU( float *in, float *out, float *padded_in, float *filters, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z, int padded_in_size_x, int padded_in_size_y, int padded_in_size_z, int padding, int kernel_size, int stride, int filter_size )
{
  int in_size = batch_size * in_size_x * in_size_y * in_size_z;
  CudaObject cuda = CudaObject();
  dim3 grid_in = cuda.cudaGridSize(in_size);
  calcConvolutionForwardPaddedInGPU<<<grid_in, BLOCK>>>(in, padded_in, in_size_x, in_size_y, in_size_z, padding);

  int out_size = batch_size * out_size_x * out_size_y * out_size_z;
  dim3 grid_out = cuda.cudaGridSize(out_size);
  calcConvolutionForwardGPU<<<grid_out, BLOCK>>>( out, padded_in, filters, padded_in_size_x, padded_in_size_y, padded_in_size_z, out_size_x, out_size_y, out_size_z, kernel_size, stride, filter_size);
}

void convolutionBockwardGPU( float *dz_next_layer, float *dz_in, float *dz, float *padded_in, int batch_size, int dz_size_x, int dz_size_y, int dz_size_z, int dz_in_size_x, int dz_in_size_y, int dz_in_size_z )
{
  int in_N = batch_size * dz_in_size_x * dz_in_size_y * dz_in_size_z;
  CudaObject cuda = CudaObject();
  dim3 grid_in = cuda.cudaGridSize(in_N);
  cudaAddFirstArrayToSecondArray<<<grid_in, BLOCK>>>( dz_next_layer, dz_in );
}

} // namespace gpu
